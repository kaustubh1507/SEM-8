// %%writefile cuda_example.cu
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// ---------------- VECTOR ADDITION ----------------
__global__ void vectorAdd(const int* A, const int* B, int* C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

// ---------------- MATRIX MULTIPLICATION ----------------
__global__ void matrixMul(const int* A, const int* B, int* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    if (row < width && col < width) {
        for (int k = 0; k < width; ++k)
            sum += A[row * width + k] * B[k * width + col];
        C[row * width + col] = sum;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << msg << " " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    // ----------------- VECTOR ADDITION -----------------
    const int N = 5;
    cout << "=== Vector Addition ===\n";
    int h_A[N] = {1, 2, 3, 4, 5};
    int h_B[N] = {10, 20, 30, 40, 50};
    int h_C[N];

    int *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc((void**)&d_A, N * sizeof(int)), "hipMalloc A failed");
    checkCudaError(hipMalloc((void**)&d_B, N * sizeof(int)), "hipMalloc B failed");
    checkCudaError(hipMalloc((void**)&d_C, N * sizeof(int)), "hipMalloc C failed");

    checkCudaError(hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice), "Memcpy A failed");
    checkCudaError(hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice), "Memcpy B failed");

    vectorAdd<<<1, N>>>(d_A, d_B, d_C, N);
    checkCudaError(hipDeviceSynchronize(), "vectorAdd failed");
    checkCudaError(hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost), "Memcpy C failed");

    cout << "A: ";
    for (int i = 0; i < N; i++) cout << h_A[i] << " ";
    cout << "\nB: ";
    for (int i = 0; i < N; i++) cout << h_B[i] << " ";
    cout << "\nC = A + B: ";
    for (int i = 0; i < N; i++) cout << h_C[i] << " ";
    cout << "\n";

    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    // ----------------- MATRIX MULTIPLICATION -----------------
    cout << "\n=== Matrix Multiplication (2x2) ===\n";
    const int WIDTH = 2;
    int h_MatA[4] = {1, 2, 3, 4}; // 2x2: [1 2; 3 4]
    int h_MatB[4] = {5, 6, 7, 8}; // 2x2: [5 6; 7 8]
    int h_MatC[4];

    int *d_MatA, *d_MatB, *d_MatC;
    checkCudaError(hipMalloc((void**)&d_MatA, 4 * sizeof(int)), "hipMalloc MatA failed");
    checkCudaError(hipMalloc((void**)&d_MatB, 4 * sizeof(int)), "hipMalloc MatB failed");
    checkCudaError(hipMalloc((void**)&d_MatC, 4 * sizeof(int)), "hipMalloc MatC failed");

    checkCudaError(hipMemcpy(d_MatA, h_MatA, 4 * sizeof(int), hipMemcpyHostToDevice), "Memcpy MatA failed");
    checkCudaError(hipMemcpy(d_MatB, h_MatB, 4 * sizeof(int), hipMemcpyHostToDevice), "Memcpy MatB failed");

    dim3 threadsPerBlock2D(2, 2);
    dim3 blocksPerGrid2D(1, 1);
    matrixMul<<<blocksPerGrid2D, threadsPerBlock2D>>>(d_MatA, d_MatB, d_MatC, WIDTH);
    checkCudaError(hipDeviceSynchronize(), "matrixMul failed");
    checkCudaError(hipMemcpy(h_MatC, d_MatC, 4 * sizeof(int), hipMemcpyDeviceToHost), "Memcpy MatC failed");

    cout << "Matrix A:\n";
    cout << h_MatA[0] << " " << h_MatA[1] << "\n" << h_MatA[2] << " " << h_MatA[3] << "\n";

    cout << "Matrix B:\n";
    cout << h_MatB[0] << " " << h_MatB[1] << "\n" << h_MatB[2] << " " << h_MatB[3] << "\n";

    cout << "Matrix C = A x B:\n";
    cout << h_MatC[0] << " " << h_MatC[1] << "\n" << h_MatC[2] << " " << h_MatC[3] << "\n";

    hipFree(d_MatA); hipFree(d_MatB); hipFree(d_MatC);
    return 0;
}